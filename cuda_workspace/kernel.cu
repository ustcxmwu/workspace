#include "hip/hip_runtime.h"
/*
 * kernel.c
 * Author:	wuxiaomin <xmwu@mail.ustc.edu.cn>
 * Date:	2013-09-17 21:39
 *
 * Copyright (C) Advanced Network Management and Control Group of USTC
 */

#include "kernel.h"

#define N 10

__global__ void add( int *a, int *b, int *c ) { 
    int tid = blockIdx.x;    // this thread handles the data at its thread id
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}


